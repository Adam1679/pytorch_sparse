#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "compat.cuh"

#define THREADS 256
#define FULL_MASK 0xffffffff

enum ReductionType { SUM, MEAN, MIN, MAX };

const std::map<std::string, ReductionType> reduce2REDUCE = {
    {"sum", SUM}, {"add", SUM}, {"mean", MEAN}, {"min", MIN}, {"max", MAX},
};

#define AT_DISPATCH_REDUCTION_TYPES(reduce, ...)                               \
  [&] {                                                                        \
    switch (reduce2REDUCE.at(reduce)) {                                        \
    case SUM: {                                                                \
      const ReductionType REDUCE = SUM;                                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case MEAN: {                                                               \
      const ReductionType REDUCE = MEAN;                                       \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case MIN: {                                                                \
      const ReductionType REDUCE = MIN;                                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case MAX: {                                                                \
      const ReductionType REDUCE = MAX;                                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    }                                                                          \
  }()

template <typename scalar_t, ReductionType REDUCE> struct Reducer {
  static inline __host__ __device__ scalar_t init() {
    if (REDUCE == MIN) {
      return std::numeric_limits<scalar_t>::max();
    } else if (REDUCE == MAX) {
      return std::numeric_limits<scalar_t>::lowest();
    } else {
      return (scalar_t)0;
    }
  }

  static inline __host__ __device__ void update(scalar_t *val, scalar_t new_val,
                                                int64_t *arg, int64_t new_arg) {
    if (REDUCE == SUM || REDUCE == MEAN) {
      *val = *val + new_val;
    } else if ((REDUCE == MIN && new_val < *val) ||
               (REDUCE == MAX && new_val > *val)) {
      *val = new_val;
      *arg = new_arg;
    }
  }

  static inline __host__ __device__ void write(scalar_t *address, scalar_t val,
                                               int64_t *arg_address,
                                               int64_t arg, int count) {
    if (REDUCE == SUM) {
      *address = val;
    } else if (REDUCE == MEAN) {
      *address = val / (scalar_t)max(count, 1);
    } else if (REDUCE == MIN || REDUCE == MAX) {
      if (count > 0) {
        *address = val;
        *arg_address = arg;
      } else {
        *address = (scalar_t)0;
      }
    }
  }
};

// Paper: Design Principles for Sparse Matrix Multiplication on the GPU
// Code:  https://github.com/owensgroup/merge-spmm
template <typename scalar_t, ReductionType REDUCE, bool HAS_VAL>
__global__ void spmm_kernel(const int64_t *rowptr_data, const int64_t *col_data,
                            const scalar_t *value_data,
                            const scalar_t *mat_data, scalar_t *out_data,
                            int64_t *arg_out_data, int B, int M, int N, int K) {

  // We ignore blockIdx.y here, because threads
  // across `blockIdx.y` are treated equally.
  int thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

  int row = thread_idx >> 5;            // thread_idx / 32
  int lane_idx = thread_idx & (32 - 1); // thread_idx % 32
  int batch_idx = row / M;

  // Compute the column index of `mat` in which the thread is operating.
  int mat_col_idx = lane_idx + (blockIdx.y << 5);

  // Compute the output index (row-major order).
  int out_idx = row * K + mat_col_idx;

  // Helper arrays for warp communication.
  int mat_row, mat_rows[32];
  scalar_t val, vals[HAS_VAL ? 32 : 1];

  // Do not aggregate/write across the Y-axis (lane_idx < leftover).
  int leftover = K - (blockIdx.y << 5);

  if (batch_idx < B) {
    int row_start = __ldg(rowptr_data + (row % M));
    int row_end = __ldg(rowptr_data + (row % M) + 1);
    int col_idx = row_start + lane_idx;

    scalar_t result = Reducer<scalar_t, REDUCE>::init();
    int64_t arg;

    // Iterate over all `col` indices in parallel within a warp.
    for (int c = row_start; c < row_end; c += 32) {

      if (col_idx < row_end) {
        // Coalesced memory access into `col` and `val`.
        mat_row = __ldg(col_data + col_idx) * K;
        if (HAS_VAL)
          val = __ldg(value_data + col_idx);
      } else {
        mat_row = -1;
        if (HAS_VAL)
          val = (scalar_t)0;
      }
      col_idx += 32;

#pragma unroll
      for (int i = 0; i < 32; i++) {
        // Communication between all threads in a warp.
        mat_rows[i] = __shfl_sync(FULL_MASK, mat_row, i);
        if (HAS_VAL)
          vals[i] = __shfl_sync(FULL_MASK, val, i);
      }

#pragma unroll
      for (int i = 0; i < 32; i++) {
        if (lane_idx < leftover && mat_rows[i] != -1) {
          // Coalesced memory access into `mat`.
          val = __ldg(mat_data + batch_idx * N * K + mat_rows[i] + mat_col_idx);
          if (HAS_VAL)
            val = vals[i] * val;
          Reducer<scalar_t, REDUCE>::update(&result, val, &arg, c + i);
        }
      }
    }

    if (lane_idx < leftover) {
      // Coalesced write into `out`.
      Reducer<scalar_t, REDUCE>::write(out_data + out_idx, result,
                                       arg_out_data + out_idx, arg,
                                       row_end - row_start);
    }
  }
}

std::tuple<torch::Tensor, torch::optional<torch::Tensor>>
spmm_cuda(torch::Tensor rowptr, torch::Tensor col,
          torch::optional<torch::Tensor> value_opt, torch::Tensor mat,
          std::string reduce) {

  AT_ASSERTM(rowptr.dim() == 1, "Input mismatch");
  AT_ASSERTM(col.dim() == 1, "Input mismatch");
  if (value_opt.has_value())
    AT_ASSERTM(value_opt.value().dim() == 1);
  AT_ASSERTM(mat.dim() >= 2, "Input mismatch");

  mat = mat.contiguous();

  auto sizes = mat.sizes().vec();
  sizes[mat.dim() - 2] = rowptr.numel() - 1;
  auto out = torch::empty(sizes, mat.options());

  torch::optional<torch::Tensor> arg_out = torch::nullopt;
  int64_t *arg_out_data = nullptr;
  if (reduce2REDUCE.at(reduce) == MIN || reduce2REDUCE.at(reduce) == MAX) {
    arg_out = torch::full_like(out, col.numel(), rowptr.options());
    arg_out_data = arg_out.value().DATA_PTR<int64_t>();
  }

  auto M = rowptr.numel() - 1;
  auto N = mat.size(-2);
  auto K = mat.size(-1);
  auto B = mat.numel() / (N * K);
  auto BLOCKS = dim3((32 * B * M + THREADS - 1) / THREADS, (K + 31) / 32);

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(mat.scalar_type(), "spmm_kernel", [&] {
    auto rowptr_data = rowptr.DATA_PTR<int64_t>();
    auto col_data = col.DATA_PTR<int64_t>();
    auto mat_data = mat.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      if (value_opt.has_value()) {
        auto value_data = value_opt.value().DATA_PTR<scalar_t>();
        spmm_kernel<scalar_t, REDUCE, true><<<BLOCKS, THREADS, 0, stream>>>(
            rowptr_data, col_data, value_data, mat_data, out_data, arg_out_data,
            B, M, N, K);
      } else {
        spmm_kernel<scalar_t, REDUCE, false><<<BLOCKS, THREADS, 0, stream>>>(
            rowptr_data, col_data, nullptr, mat_data, out_data, arg_out_data, B,
            M, N, K);
      }
    });
  });

  return std::make_tuple(out, arg_out);
}

template <typename scalar_t, ReductionType REDUCE>
__global__ void
spmm_val_bw_kernel(const int64_t *row_data, const int64_t *rowptr_data,
                   const int64_t *col_data, const scalar_t *mat_data,
                   const scalar_t *grad_data, scalar_t *out_data, int B, int M,
                   int N, int E, int K) {
  int thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

  int index_idx = (thread_idx >> 5);    // thread_idx / 32
  int lane_idx = thread_idx & (32 - 1); // thread_idx % 32

  if (index_idx < E) {
    int row = __ldg(row_data + index_idx);
    int col = __ldg(col_data + index_idx);

    scalar_t val = (scalar_t)0;
    for (int b = 0; b < B; b++) {
      for (int k = lane_idx; k < K; k += 32) {
        val += mat_data[b * N * K + col * K + k] *
               grad_data[b * M * K + row * K + k];
      }
    }

#pragma unroll
    for (int i = 32 / 2; i > 0; i /= 2) { // Parallel reduction inside a warp.
      val += __shfl_down_sync(FULL_MASK, val, i);
    }

    if (lane_idx == 0) {
      if (REDUCE == MEAN) {
        int row_start = __ldg(rowptr_data + row);
        int row_end = __ldg(rowptr_data + row + 1);
        val /= (scalar_t)max(row_end - row_start, 1);
      }
      out_data[index_idx] = val;
    }
  }
}

torch::Tensor spmm_val_bw_cuda(torch::Tensor row, torch::Tensor rowptr,
                               torch::Tensor col, torch::Tensor mat,
                               torch::Tensor grad, std::string reduce) {

  mat = mat.contiguous();
  grad = grad.contiguous();

  auto M = grad.size(-2);
  auto N = mat.size(-2);
  auto E = row.numel();
  auto K = mat.size(-1);
  auto B = mat.numel() / (N * K);
  auto BLOCKS = dim3((E * 32 + THREADS - 1) / THREADS);

  auto out = torch::zeros(row.numel(), grad.options());

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(mat.scalar_type(), "spmm_val_bw_kernel", [&] {
    auto row_data = row.DATA_PTR<int64_t>();
    auto rowptr_data = rowptr.DATA_PTR<int64_t>();
    auto col_data = col.DATA_PTR<int64_t>();
    auto mat_data = mat.DATA_PTR<scalar_t>();
    auto grad_data = grad.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      spmm_val_bw_kernel<scalar_t, REDUCE><<<BLOCKS, THREADS, 0, stream>>>(
          row_data, rowptr_data, col_data, mat_data, grad_data, out_data, B, M,
          N, E, K);
    });
  });

  return out;
}
