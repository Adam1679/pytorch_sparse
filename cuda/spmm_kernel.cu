#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include "compat.cuh"

#define THREADS 256
#define FULL_MASK 0xffffffff

enum ReductionType { SUM, MEAN, MIN, MAX };

const std::map<std::string, ReductionType> reduce2REDUCE = {
    {"sum", SUM}, {"add", SUM}, {"mean", MEAN}, {"min", MIN}, {"max", MAX},
};

#define AT_DISPATCH_REDUCTION_TYPES(reduce, ...)                               \
  [&] {                                                                        \
    switch (reduce2REDUCE.at(reduce)) {                                        \
    case SUM: {                                                                \
      const ReductionType REDUCE = SUM;                                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case MEAN: {                                                               \
      const ReductionType REDUCE = MEAN;                                       \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case MIN: {                                                                \
      const ReductionType REDUCE = MIN;                                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case MAX: {                                                                \
      const ReductionType REDUCE = MAX;                                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    }                                                                          \
  }()

template <typename scalar_t, ReductionType REDUCE> struct Reducer {
  static inline __host__ __device__ scalar_t init() {
    if (REDUCE == MIN) {
      return std::numeric_limits<scalar_t>::max();
    } else if (REDUCE == MAX) {
      return std::numeric_limits<scalar_t>::lowest();
    } else {
      return (scalar_t)0;
    }
  }

  static inline __host__ __device__ void update(scalar_t *val, scalar_t new_val,
                                                int64_t *arg, int64_t new_arg) {
    if (REDUCE == SUM || REDUCE == MEAN) {
      *val = *val + new_val;
    } else if ((REDUCE == MIN && new_val < *val) ||
               (REDUCE == MAX && new_val > *val)) {
      *val = new_val;
      *arg = new_arg;
    }
  }

  static inline __host__ __device__ void write(scalar_t *address, scalar_t val,
                                               int64_t *arg_address,
                                               int64_t arg, int count) {
    if (REDUCE == SUM) {
      *address = val;
    } else if (REDUCE == MEAN) {
      *address = val / (scalar_t)max(count, 1);
    } else if (REDUCE == MIN || REDUCE == MAX) {
      if (count > 0) {
        *address = val;
        *arg_address = arg;
      } else {
        *address = (scalar_t)0;
      }
    }
  }
};

// Paper: Design Principles for Sparse Matrix Multiplication on the GPU
// Code:  https://github.com/owensgroup/merge-spmm
template <typename scalar_t, ReductionType REDUCE, bool HAS_VAL>
__global__ void spmm_kernel(const int64_t *rowptr_data, const int64_t *col_data,
                            const scalar_t *value_data,
                            const scalar_t *mat_data, scalar_t *out_data,
                            int64_t *arg_out_data, int B, int M, int N, int K) {

  // We ignore blockIdx.y here, because threads
  // across `blockIdx.y` are treated equally.
  int thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

  int row = thread_idx >> 5;            // thread_idx / 32
  int lane_idx = thread_idx & (32 - 1); // thread_idx % 32
  int batch_idx = row / M;

  // Compute the column index of `mat` in which the thread is operating.
  int mat_col_idx = lane_idx + (blockIdx.y << 5);

  // Compute the output index (row-major order).
  int out_idx = row * K + mat_col_idx;

  // Helper arrays for warp communication.
  int mat_row, mat_rows[32];
  scalar_t val, vals[HAS_VAL ? 32 : 1];
  int bla, blas[32];

  // Do not aggregate/write across the Y-axis (lane_idx < leftover).
  int leftover = K - (blockIdx.y << 5);

  if (row < B * M) {
    int row_start = __ldg(rowptr_data + (row % M));
    int row_end = __ldg(rowptr_data + (row % M) + 1);
    int col_idx = row_start + lane_idx;

    scalar_t result = Reducer<scalar_t, REDUCE>::init();
    int64_t arg;

    // Iterate over all `col` indices in parallel within a warp.
    for (int c = row_start; c < row_end; c += 32) {

      if (col_idx < row_end) {
        // Coalesced memory access into `col` and `val`.
        mat_row = __ldg(col_data + col_idx) * K;
        bla = col_idx;
        if (HAS_VAL)
          val = __ldg(value_data + col_idx);
      } else {
        mat_row = -1;
        bla = -1;
        if (HAS_VAL)
          val = (scalar_t)0;
      }
      col_idx += 32;

#pragma unroll
      for (int i = 0; i < 32; i++) {
        // Communication between all threads in a warp.
        mat_rows[i] = __shfl_sync(FULL_MASK, mat_row, i);
        blas[i] = __shfl_sync(FULL_MASK, bla, i);
        if (HAS_VAL)
          vals[i] = __shfl_sync(FULL_MASK, val, i);
      }

#pragma unroll
      for (int i = 0; i < 32; i++) {
        if (lane_idx < leftover && mat_rows[i] != -1) {
          // Coalesced memory access into `mat`.
          val = __ldg(mat_data + batch_idx * N * K + mat_rows[i] + mat_col_idx);
          if (HAS_VAL)
            val = vals[i] * val;
          Reducer<scalar_t, REDUCE>::update(&result, val, &arg, c + i);
        }
      }
    }

    if (lane_idx < leftover) {
      // Coalesced write into `out`.
      Reducer<scalar_t, REDUCE>::write(out_data + out_idx, result,
                                       arg_out_data + out_idx, arg,
                                       row_end - row_start);
    }
  }
}

std::tuple<at::Tensor, at::optional<at::Tensor>>
spmm_cuda(at::Tensor rowptr, at::Tensor col, at::optional<at::Tensor> value_opt,
          at::Tensor mat, std::string reduce) {

  AT_ASSERTM(rowptr.dim() == 1, "Input mismatch");
  AT_ASSERTM(col.dim() == 1, "Input mismatch");
  if (value_opt.has_value())
    AT_ASSERTM(value_opt.value().dim() == 1);
  AT_ASSERTM(mat.dim() >= 2, "Input mismatch");

  mat = mat.contiguous();

  auto sizes = mat.sizes().vec();
  sizes[mat.dim() - 2] = rowptr.numel() - 1;
  auto out = at::empty(sizes, mat.options());

  at::optional<at::Tensor> arg_out = at::nullopt;
  int64_t *arg_out_data = nullptr;
  if (reduce2REDUCE.at(reduce) == MIN || reduce2REDUCE.at(reduce) == MAX) {
    arg_out = at::full_like(out, col.numel(), rowptr.options());
    arg_out_data = arg_out.value().DATA_PTR<int64_t>();
  }

  auto rowptr_data = rowptr.DATA_PTR<int64_t>();
  auto col_data = col.DATA_PTR<int64_t>();

  auto M = rowptr.numel() - 1;
  auto N = mat.size(-2);
  auto K = mat.size(-1);
  auto B = mat.numel() / (N * K);
  auto BLOCKS = dim3((32 * B * M + THREADS - 1) / THREADS, (K + 31) / 32);

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(mat.scalar_type(), "spmm_kernel", [&] {
    auto mat_data = mat.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      if (value_opt.has_value()) {
        auto value_data = value_opt.value().DATA_PTR<scalar_t>();
        spmm_kernel<scalar_t, REDUCE, true><<<BLOCKS, THREADS, 0, stream>>>(
            rowptr_data, col_data, value_data, mat_data, out_data, arg_out_data,
            B, M, N, K);
      } else {
        spmm_kernel<scalar_t, REDUCE, false><<<BLOCKS, THREADS, 0, stream>>>(
            rowptr_data, col_data, nullptr, mat_data, out_data, arg_out_data, B,
            M, N, K);
      }
    });
  });

  return std::make_tuple(out, arg_out);
}
