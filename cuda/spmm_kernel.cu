#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include "compat.cuh"

#define THREADS 256

#define ADD 0
#define MEAN 1
#define MIN 2
#define MAX 3

// Paper: Design Principles for Sparse Matrix Multiplication on the GPU
// Code:  https://github.com/owensgroup/merge-spmm
template <typename scalar_t, int64_t REDUCE, bool HAS_VAL>
__global__ void spmm_kernel(const int64_t *rowptr_data, const int64_t *col_data,
                            const scalar_t *val_data, const scalar_t *mat_data,
                            scalar_t *out_data, int64_t *arg_out_data, size_t N,
                            size_t K) {

  // We ignore blockIdx.y here, because threads
  // across `blockIdx.y` are treated equally.
  int thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

  int row = thread_idx >> 5;            // thread_id / 32
  int lane_idx = thread_idx & (32 - 1); // thread_id % 32

  // Compute the column index of `mat` in which the thread is operating.
  int mat_col_idx = lane_idx + (blockIdx.y << 5);

  // Compute the output index (row-major order).
  int out_idx = row * K + lane_idx + (blockIdx.y << 5);

  // Helper arrays for warp communication.
  int mat_rows[32];
  scalar_t vals[32];

  // Do not aggregate/write across the Y-axis (lane_idx < leftover).
  int leftover = K - (blockIdx.y << 5);

  if (row < N) {
    int row_start = __ldg(rowptr_data + row);
    int row_end = __ldg(rowptr_data + row + 1);
    int col_idx = row_start + lane_idx;

    int mat_row;
    scalar_t val, result;
    int64_t arg_result = -1;

    // Dependent on `reduce`, we need to initialize `result` accordingly.
    if (REDUCE == ADD)
      result = (scalar_t)0;
    else if (REDUCE == MEAN)
      result = (scalar_t)0;
    else if (REDUCE == MIN)
      result = std::numeric_limits<scalar_t>::max();
    else if (REDUCE == MAX)
      result = std::numeric_limits<scalar_t>::min();

    // Iterate over all col indices in parallel within a warp.
    for (int c = row_start; c < row_end; c += 32) {

      if (col_idx < row_end) {
        // Coalesced memory access into `col` and `val`.
        mat_row = __ldg(col_data + col_idx) * K;
        val = HAS_VAL ? __ldg(val_data + col_idx) : (scalar_t)1;
      } else {
        mat_row = 0;
        val = (scalar_t)0;
      }
      col_idx += 32;

#pragma unroll
      for (int i = 0; i < 32; i++) {
        // Communication between all threads in a warp.
        mat_rows[i] = __shfl_sync(0xffffffff, mat_row, i);
        vals[i] = __shfl_sync(0xffffffff, val, i);
      }

#pragma unroll
      for (int i = 0; i < 32; i++) {
        if (lane_idx < leftover && vals[i] != 0) {
          // Coalesced memory access into `mat`.
          val = vals[i] * __ldg(mat_data + mat_rows[i] + mat_col_idx);

          // Aggregate results along row.
          if (REDUCE == ADD)
            result += val;
          else if (REDUCE == MEAN)
            result += val;
          else if (REDUCE == MIN) {
            if (val < result) {
              result = val;
              arg_result = row_start + i;
            }
          } else if (REDUCE == MAX) {
            if (val > result) {
              result = val;
              arg_result = row_start + i;
            }
          }
        }
      }
    }

    if (lane_idx < leftover) {
      // Coalesced write into `out` (dependent on `reduce`).
      if (REDUCE == ADD)
        out_data[out_idx] = result;
      else if (REDUCE == MEAN)
        out_data[out_idx] = result / scalar_t(row_end - row_start);
      else if (REDUCE == MIN) {
        arg_out_data[out_idx] = arg_result;
        if (result == std::numeric_limits<scalar_t>::max())
          out_data[out_idx] = (scalar_t)0;
        else
          out_data[out_idx] = result;
      } else if (REDUCE == MAX) {
        arg_out_data[out_idx] = arg_result;
        if (result == std::numeric_limits<scalar_t>::min())
          out_data[out_idx] = (scalar_t)0;
        else
          out_data[out_idx] = result;
      }
    }
  }
}

at::Tensor spmm_cuda(at::Tensor rowptr, at::Tensor col,
                     at::optional<at::Tensor> val, at::Tensor mat,
                     std::string reduce) {

  auto N = rowptr.size(0) - 1;
  auto K = mat.size(1);
  auto out = at::empty({N, K}, mat.options());

  auto rowptr_data = rowptr.DATA_PTR<int64_t>();
  auto col_data = col.DATA_PTR<int64_t>();

  auto block = dim3(THREADS);
  auto grid = dim3((32 * N + THREADS - 1) / THREADS, (K + 31) / 32);

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(mat.scalar_type(), "spmm_kernel", [&] {
    auto mat_data = mat.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    if (val.has_value()) {
      auto val_data = val.value().DATA_PTR<scalar_t>();
      if (reduce == "add")
        spmm_kernel<scalar_t, ADD, true><<<grid, block, 0, stream>>>(
            rowptr_data, col_data, val_data, mat_data, out_data, nullptr, N, K);
      else if (reduce == "mean")
        spmm_kernel<scalar_t, MEAN, true><<<grid, block, 0, stream>>>(
            rowptr_data, col_data, val_data, mat_data, out_data, nullptr, N, K);
    } else {
      if (reduce == "add")
        spmm_kernel<scalar_t, ADD, false><<<grid, block, 0, stream>>>(
            rowptr_data, col_data, nullptr, mat_data, out_data, nullptr, N, K);
      else if (reduce == "mean")
        spmm_kernel<scalar_t, MEAN, false><<<grid, block, 0, stream>>>(
            rowptr_data, col_data, nullptr, mat_data, out_data, nullptr, N, K);
    }
  });

  return out;
}

std::tuple<at::Tensor, at::Tensor>
spmm_arg_cuda(at::Tensor rowptr, at::Tensor col, at::optional<at::Tensor> val,
              at::Tensor mat, std::string reduce) {

  auto N = rowptr.size(0) - 1;
  auto K = mat.size(1);
  auto out = at::empty({N, K}, mat.options());
  auto arg_out = at::empty({N, K}, rowptr.options());

  auto rowptr_data = rowptr.DATA_PTR<int64_t>();
  auto col_data = col.DATA_PTR<int64_t>();
  auto arg_out_data = arg_out.DATA_PTR<int64_t>();

  auto block = dim3(THREADS);
  auto grid = dim3((32 * N + THREADS - 1) / THREADS, (K + 31) / 32);

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(mat.scalar_type(), "spmm_kernel", [&] {
    auto mat_data = mat.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    if (val.has_value()) {
      auto val_data = val.value().DATA_PTR<scalar_t>();
      if (reduce == "min")
        spmm_kernel<scalar_t, MIN, true><<<grid, block, 0, stream>>>(
            rowptr_data, col_data, val_data, mat_data, out_data, arg_out_data,
            N, K);
      else if (reduce == "max")
        spmm_kernel<scalar_t, MAX, true><<<grid, block, 0, stream>>>(
            rowptr_data, col_data, val_data, mat_data, out_data, arg_out_data,
            N, K);
    } else {
      if (reduce == "min")
        spmm_kernel<scalar_t, MIN, false><<<grid, block, 0, stream>>>(
            rowptr_data, col_data, nullptr, mat_data, out_data, arg_out_data, N,
            K);
      else if (reduce == "max")
        spmm_kernel<scalar_t, MAX, false><<<grid, block, 0, stream>>>(
            rowptr_data, col_data, nullptr, mat_data, out_data, arg_out_data, N,
            K);
    }
  });

  return std::make_tuple(out, arg_out);
}
