#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include "compat.cuh"

#define THREADS 32 * 16

// Paper: Design Principles for Sparse Matrix Multiplication on the GPU
// Code:  https://github.com/owensgroup/merge-spmm
template <typename scalar_t, size_t Y_SIZE>
__global__ void
spmm_row_kernel(const int64_t *rowptr_data, const int64_t *col_data,
                const scalar_t *val_data, const scalar_t *mat_data,
                scalar_t *out_data, size_t N, size_t M, size_t K) {

  // We ignore blockIdx.y here, because threads across blockIdx.y operate on the
  // same row.
  int thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

  int warp_idx = thread_idx >> 5;       // thread_id / 32
  int lane_idx = thread_idx & (32 - 1); // thread_id % 32
  int row = warp_idx;                   // Each warp processes exactly one row.

  // Compute the column index of `mat` in which the thread is operating.
  int mat_col_idx = lane_idx + (blockIdx.y << 5);

  // Compute the output index given in row-major order.
  int out_idx = row * K + lane_idx + (blockIdx.y << 5);

  // Helper arrays for warp communication.
  int mat_row_all[Y_SIZE];
  scalar_t val_all[Y_SIZE];

  int leftover = K - (blockIdx.y << 5);

  if (row < N) {
    int row_start = __ldg(rowptr_data + row);
    int row_end = __ldg(rowptr_data + row + 1);

    // Iterate over all col indices in parallel.
    for (int col_idx = row_start + lane_idx; col_idx < row_end; col_idx += 32) {
      int mat_row = __ldg(col_data + col_idx) * K;
      int val = __ldg(val_data + col_idx);
      scalar_t sum = (scalar_t)0;

      for (int i = 0; i < 32; i += Y_SIZE) {
#pragma unroll
        for (int j = 0; j < Y_SIZE; j++) {
          // Warp communication with *all* threads (mask = 0xffffffff).
          // TODO: Compute real bit mask via `__ballot_sync()`.
          mat_row_all[j] = __shfl_sync(0xffffffff, mat_row, i + j);
          val_all[j] = __shfl_sync(0xffffffff, val, i + j);
        }
#pragma unroll
        for (int j = 0; j < Y_SIZE; j++) {
          if (lane_idx < leftover) {
            // Coalesced memory access into `mat`.
            sum += val_all[j] * __ldg(mat_data + mat_row_all[j] + mat_col_idx);
          }
        }
      }
      if (lane_idx < leftover) {
        out_data[out_idx] = sum;
      }
    }
  }
}

at::Tensor spmm_cuda(at::Tensor rowptr, at::Tensor col, at::Tensor val,
                     at::Tensor mat) {
  // TODO: Set device

  auto N = rowptr.numel() - 1;
  auto M = mat.size(0);
  auto K = mat.size(1);

  auto out = at::empty({N, K}, mat.options());

  auto rowptr_data = rowptr.DATA_PTR<int64_t>();
  auto col_data = col.DATA_PTR<int64_t>();
  auto val_data = val.DATA_PTR<float>();
  auto mat_data = mat.DATA_PTR<float>();
  auto out_data = out.DATA_PTR<float>();

  auto block_dim = dim3(THREADS);
  auto grid_dim = dim3((N + THREADS - 1) / THREADS, (K + 32 - 1) / 32);

  spmm_row_kernel<float, 32><<<grid_dim, block_dim, 0 /*, hip_stream */>>>(
      rowptr_data, col_data, val_data, mat_data, out_data, N, M, K);

  return out;
}
