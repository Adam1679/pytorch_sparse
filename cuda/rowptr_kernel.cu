#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include "compat.cuh"

#define THREADS 256

__global__ void rowptr_kernel(const int64_t *row_data, int64_t *out_data,
                              int64_t numel, int64_t size) {

  int64_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (thread_idx == 0) {
    for (int64_t i = 0; i < row_data[0]; i++)
      out_data[i] = 0;
  } else if (thread_idx == numel) {
    for (int64_t i = row_data[numel - 1]; i < size + 1; i++)
      out_data[i] = size;
  } else {
    for (int64_t i = row_data[thread_idx - 1]; i < row_data[thread_idx]; i++)
      out_data[i] = thread_idx - 1;
  }
}

at::Tensor rowptr_cuda(at::Tensor row, size_t size) {
  AT_ASSERTM(row.dim() == 1, "Row needs to be one-dimensional");

  auto out = at::empty(size + 1, row.options());
  auto row_data = row.DATA_PTR<int64_t>();
  auto out_data = out.DATA_PTR<int64_t>();

  auto stream = at::cuda::getCurrentCUDAStream();
  rowptr_kernel<<<(row.numel() + 2 + THREADS - 1) / THREADS, THREADS, 0,
                  stream>>>(row_data, out_data, row.numel(), size);

  return out;
}
