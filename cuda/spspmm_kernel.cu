#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <hipsparse.h>

#include "compat.cuh"

#define AT_DISPATCH_CUSPARSE_CSR_GEMM2_BUFFER_SIZE_EXT_TYPES(TYPE, ...)        \
  [&] {                                                                        \
    switch (TYPE) {                                                            \
    case at::ScalarType::Float: {                                              \
      using scalar_t = float;                                                  \
      const auto &cusparsecsrgemm2_bufferSizeExt =                             \
          hipsparseScsrgemm2_bufferSizeExt;                                     \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case at::ScalarType::Double: {                                             \
      using scalar_t = double;                                                 \
      const auto &cusparsecsrgemm2_bufferSizeExt =                             \
          hipsparseDcsrgemm2_bufferSizeExt;                                     \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    default:                                                                   \
      AT_ERROR("Not implemented for '", toString(TYPE), "'");                  \
    }                                                                          \
  }()

#define AT_DISPATCH_CUSPARSE_CSR_GEMM2_TYPES(TYPE, ...)                        \
  [&] {                                                                        \
    switch (TYPE) {                                                            \
    case at::ScalarType::Float: {                                              \
      using scalar_t = float;                                                  \
      const auto &cusparsecsrgemm2 = hipsparseScsrgemm2;                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case at::ScalarType::Double: {                                             \
      using scalar_t = double;                                                 \
      const auto &cusparsecsrgemm2 = hipsparseDcsrgemm2;                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    default:                                                                   \
      AT_ERROR("Not implemented for '", toString(TYPE), "'");                  \
    }                                                                          \
  }()

std::tuple<at::Tensor, at::Tensor, at::optional<at::Tensor>>
spspmm_cuda(at::Tensor rowptrA, at::Tensor colA,
            at::optional<at::Tensor> valueA, at::Tensor rowptrB,
            at::Tensor colB, at::optional<at::Tensor> valueB, int M, int N,
            int K) {
  hipsparseMatDescr_t descr = 0;
  hipsparseCreateMatDescr(&descr);
  auto handle = at::cuda::getCurrentCUDASparseHandle();

  rowptrA = rowptrA.toType(at::kInt), colA = colA.toType(at::kInt);
  rowptrB = rowptrB.toType(at::kInt), colB = colB.toType(at::kInt);

  auto rowptrA_data = rowptrA.DATA_PTR<int>(), colA_data = colA.DATA_PTR<int>();
  auto rowptrB_data = rowptrB.DATA_PTR<int>(), colB_data = colB.DATA_PTR<int>();

  csrgemm2Info_t info = NULL;
  hipsparseCreateCsrgemm2Info(&info);

  auto scalar_type = at::ScalarType::Float;
  if (valueA.has_value())
    scalar_type = valueA.value().scalar_type();
  if (valueB.has_value())
    scalar_type = valueB.value().scalar_type();

  size_t bufferSize;
  AT_DISPATCH_CUSPARSE_CSR_GEMM2_BUFFER_SIZE_EXT_TYPES(scalar_type, [&] {
    scalar_t alpha = (scalar_t)1;
    cusparsecsrgemm2_bufferSizeExt(handle, M, N, K, &alpha, descr, colA.numel(),
                                   rowptrA_data, colA_data, descr, colB.numel(),
                                   rowptrB_data, colB_data, NULL, descr, 0,
                                   NULL, NULL, info, &bufferSize);
  });

  void *buffer = NULL;
  hipMalloc(&buffer, bufferSize);

  int nnzC;
  auto rowptrC = at::empty(M + 1, rowptrA.options());
  auto rowptrC_data = rowptrC.DATA_PTR<int>();
  hipsparseXcsrgemm2Nnz(handle, M, N, K, descr, colA.numel(), rowptrA_data,
                       colA_data, descr, colB.numel(), rowptrB_data, colB_data,
                       descr, 0, NULL, NULL, descr, rowptrC_data, &nnzC, info,
                       buffer);

  auto colC = at::empty(nnzC, colA.options());
  auto colC_data = colC.DATA_PTR<int>();

  if (!valueA.has_value() && valueB.has_value())
    valueA = at::ones_like(valueB.value());

  if (!valueB.has_value() && valueA.has_value())
    valueB = at::ones_like(valueA.value());

  at::optional<at::Tensor> valueC = at::nullopt;
  if (valueA.has_value())
    valueC = at::empty(nnzC, valueA.value().options());

  AT_DISPATCH_CUSPARSE_CSR_GEMM2_TYPES(scalar_type, [&] {
    scalar_t alpha = (scalar_t)1;

    scalar_t *valueA_data = NULL;
    if (valueA.has_value())
      valueA_data = valueA.value().DATA_PTR<scalar_t>();

    scalar_t *valueB_data = NULL;
    if (valueB.has_value())
      valueB_data = valueB.value().DATA_PTR<scalar_t>();

    scalar_t *valueC_data = NULL;
    if (valueC.has_value())
      valueC_data = valueC.value().DATA_PTR<scalar_t>();

    cusparsecsrgemm2(handle, M, N, K, &alpha, descr, colA.numel(), valueA_data,
                     rowptrA_data, colA_data, descr, colB.numel(), valueB_data,
                     rowptrB_data, colB_data, NULL, descr, 0, NULL, NULL, NULL,
                     descr, valueC_data, rowptrC_data, colC_data, info, buffer);
  });

  rowptrC = rowptrC.toType(at::kLong);
  colC = colC.toType(at::kLong);

  return std::make_tuple(rowptrC, colC, valueC);
}

// #define THREADS 1024
// #define BLOCKS(N) (N + THREADS - 1) / THREADS

// #define CSRGEMM(TYPE, ...)                                                     \
//   [&] {                                                                        \
//     const auto &the_type = TYPE;                                               \
//     (void)the_type;                                                            \
//     at::ScalarType _st = ::detail::scalar_type(TYPE);                          \
//     switch (_st) {                                                             \
//     case at::ScalarType::Float: {                                              \
//       using scalar_t = float;                                                  \
//       return hipsparseScsrgemm(__VA_ARGS__);                                    \
//     }                                                                          \
//     case at::ScalarType::Double: {                                             \
//       using scalar_t = double;                                                 \
//       return hipsparseDcsrgemm(__VA_ARGS__);                                    \
//     }                                                                          \
//     default:                                                                   \
//       AT_ERROR("Not implemented for '", toString(_st), "'");                   \
//     }                                                                          \
//   }()

// static hipsparseHandle_t cusparse_handle = 0;

// static void init_cusparse() {
//   if (cusparse_handle == 0) {
//     hipsparseStatus_t status = hipsparseCreate(&cusparse_handle);
//   }
// }

// std::tuple<at::Tensor, at::Tensor>
// spspmm_cuda(at::Tensor indexA, at::Tensor valueA, at::Tensor indexB,
//             at::Tensor valueB, size_t m, size_t k, size_t n) {
//   hipSetDevice(indexA.get_device());
//   init_cusparse();

//   indexA = indexA.contiguous();
//   valueA = valueA.contiguous();
//   indexB = indexB.contiguous();
//   valueB = valueB.contiguous();

//   auto nnzA = valueA.size(0);
//   auto nnzB = valueB.size(0);

//   indexA = indexA.toType(at::kInt);
//   indexB = indexB.toType(at::kInt);

//   // Convert A to CSR format.
//   auto row_ptrA = at::empty(m + 1, indexA.options());
//   hipsparseXcoo2csr(cusparse_handle, indexA[0].DATA_PTR<int>(), nnzA, k,
//                    row_ptrA.DATA_PTR<int>(), HIPSPARSE_INDEX_BASE_ZERO);
//   auto colA = indexA[1];
//   hipMemcpy(row_ptrA.DATA_PTR<int>() + m, &nnzA, sizeof(int),
//              hipMemcpyHostToDevice);

//   // Convert B to CSR format.
//   auto row_ptrB = at::empty(k + 1, indexB.options());
//   hipsparseXcoo2csr(cusparse_handle, indexB[0].DATA_PTR<int>(), nnzB, k,
//                    row_ptrB.DATA_PTR<int>(), HIPSPARSE_INDEX_BASE_ZERO);
//   auto colB = indexB[1];
//   hipMemcpy(row_ptrB.DATA_PTR<int>() + k, &nnzB, sizeof(int),
//              hipMemcpyHostToDevice);

//   hipsparseMatDescr_t descr = 0;
//   hipsparseCreateMatDescr(&descr);
//   hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
//   hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

//   int nnzC;
//   auto row_ptrC = at::empty(m + 1, indexB.options());
//   hipsparseXcsrgemmNnz(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//                       HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr,
//                       nnzA, row_ptrA.DATA_PTR<int>(),
//                       colA.DATA_PTR<int>(), descr, nnzB,
//                       row_ptrB.DATA_PTR<int>(), colB.DATA_PTR<int>(),
//                       descr, row_ptrC.DATA_PTR<int>(), &nnzC);
//   auto colC = at::empty(nnzC, indexA.options());
//   auto valueC = at::empty(nnzC, valueA.options());

//   CSRGEMM(valueC.scalar_type(), cusparse_handle,
//           HIPSPARSE_OPERATION_NON_TRANSPOSE,
//           HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnzA,
//           valueA.DATA_PTR<scalar_t>(), row_ptrA.DATA_PTR<int>(),
//           colA.DATA_PTR<int>(), descr, nnzB, valueB.DATA_PTR<scalar_t>(),
//           row_ptrB.DATA_PTR<int>(), colB.DATA_PTR<int>(), descr,
//           valueC.DATA_PTR<scalar_t>(), row_ptrC.DATA_PTR<int>(),
//           colC.DATA_PTR<int>());

//   auto rowC = at::empty(nnzC, indexA.options());
//   hipsparseXcsr2coo(cusparse_handle, row_ptrC.DATA_PTR<int>(), nnzC, m,
//                    rowC.DATA_PTR<int>(), HIPSPARSE_INDEX_BASE_ZERO);

//   auto indexC = at::stack({rowC, colC}, 0).toType(at::kLong);

//   return std::make_tuple(indexC, valueC);
// }

// at::Tensor degree(at::Tensor row, int64_t num_nodes) {
//   auto zero = at::zeros(num_nodes, row.options());
//   auto one = at::ones(row.size(0), row.options());
//   return zero.scatter_add_(0, row, one);
// }

// std::tuple<at::Tensor, at::Tensor> to_csr(at::Tensor row, at::Tensor col,
//                                           int64_t num_nodes) {
//   // Assert already coalesced input.
//   row = degree(row, num_nodes).cumsum(0);
//   row = at::cat({at::zeros(1, row.options()), row}, 0); // Prepend zero.
//   return std::make_tuple(row, col);
// }

// template <typename scalar_t>
// __global__ void spspmm_bw_kernel(
//     const int64_t *__restrict__ index, scalar_t *__restrict__ value,
//     const int64_t *__restrict__ rowA, const int64_t *__restrict__ colA,
//     const scalar_t *__restrict__ valueA, const int64_t *__restrict__
//     rowB, const int64_t *__restrict__ colB, const scalar_t *__restrict__
//     valueB, const size_t numel) {
//   const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
//   const size_t stride = blockDim.x * gridDim.x;
//   for (ptrdiff_t e = idx; e < numel; e += stride) {
//     int64_t i = index[e], j = index[numel + e];

//     for (ptrdiff_t dA = rowA[i]; dA < rowA[i + 1]; dA++) {
//       int64_t cA = colA[dA];

//       for (ptrdiff_t dB = rowB[j]; dB < rowB[j + 1]; dB++) {
//         int64_t cB = colB[dB];

//         if (cA == cB) {
//           value[e] += valueA[dA] * valueB[dB];
//         }

//         if (cB >= cA) {
//           break;
//         }
//       }
//     }
//   }
// }

// at::Tensor spspmm_bw_cuda(at::Tensor index, at::Tensor indexA,
//                           at::Tensor valueA, at::Tensor indexB,
//                           at::Tensor valueB, size_t rowA_max, size_t
//                           rowB_max) {
//   hipSetDevice(index.get_device());
//   auto value = at::zeros(index.size(1), valueA.options());

//   at::Tensor rowA, colA;
//   std::tie(rowA, colA) = to_csr(indexA[0], indexA[1], rowA_max);

//   at::Tensor rowB, colB;
//   std::tie(rowB, colB) = to_csr(indexB[0], indexB[1], rowB_max);

//   AT_DISPATCH_FLOATING_TYPES(valueA.scalar_type(), "spspmm_bw", [&] {
//     spspmm_bw_kernel<scalar_t><<<BLOCKS(value.numel()), THREADS>>>(
//         index.DATA_PTR<int64_t>(), value.DATA_PTR<scalar_t>(),
//         rowA.DATA_PTR<int64_t>(), colA.DATA_PTR<int64_t>(),
//         valueA.DATA_PTR<scalar_t>(), rowB.DATA_PTR<int64_t>(),
//         colB.DATA_PTR<int64_t>(), valueB.DATA_PTR<scalar_t>(),
//         value.numel());
//   });

//   return value;
// }
